#include "hip/hip_runtime.h"
#include <assert.h>
#include "gpu-utils/device_context.cuh"
#include "gpu-utils/error_handler.cuh"
#include "gpu-utils/modifiers.cuh"
#include "hash/hash.cuh"

#include "hash/blake2s/blake2s.cuh"
using namespace hash;

#include "matrix/matrix.cuh"
using matrix::Matrix;

namespace blake2s {

  typedef struct {
    WORD digestlen;
    BYTE key[32];
    WORD keylen;
    BYTE buff[BLAKE2S_BLOCK_LENGTH];
    uint32_t chain[BLAKE2S_CHAIN_SIZE];
    uint32_t state[BLAKE2S_STATE_SIZE];
    WORD pos;
    uint32_t t0;
    uint32_t t1;
    uint32_t f0;
  } cuda_blake2s_ctx_t;

  typedef cuda_blake2s_ctx_t CUDA_BLAKE2S_CTX;

  __constant__ CUDA_BLAKE2S_CTX c_CTX;

  __constant__ uint32_t BLAKE2S_IVS[8] = {0x6A09E667UL, 0xBB67AE85UL, 0x3C6EF372UL, 0xA54FF53AUL,
                                          0x510E527FUL, 0x9B05688CUL, 0x1F83D9ABUL, 0x5BE0CD19UL};

  const uint32_t CPU_BLAKE2S_IVS[8] = {0x6A09E667UL, 0xBB67AE85UL, 0x3C6EF372UL, 0xA54FF53AUL,
                                       0x510E527FUL, 0x9B05688CUL, 0x1F83D9ABUL, 0x5BE0CD19UL};

  static DEVICE_INLINE unsigned int d_next_pow_of_two(unsigned int v)
  {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
  }

  void cpu_blake2s_init(cuda_blake2s_ctx_t* ctx, BYTE* key, WORD keylen, WORD digestbitlen)
  {
    memset(ctx, 0, sizeof(cuda_blake2s_ctx_t));
    if (keylen > 0) {
      memcpy(ctx->buff, key, keylen);
      memcpy(ctx->key, key, keylen);
    }
    ctx->keylen = keylen;
    ctx->digestlen = digestbitlen >> 3;
    ctx->pos = 0;
    ctx->t0 = 0;
    ctx->t1 = 0;
    ctx->f0 = 0;
    ctx->chain[0] = CPU_BLAKE2S_IVS[0] ^ (ctx->digestlen | (ctx->keylen << 8) | 0x1010000);
    ctx->chain[1] = CPU_BLAKE2S_IVS[1];
    ctx->chain[2] = CPU_BLAKE2S_IVS[2];
    ctx->chain[3] = CPU_BLAKE2S_IVS[3];
    ctx->chain[4] = CPU_BLAKE2S_IVS[4];
    ctx->chain[5] = CPU_BLAKE2S_IVS[5];
    ctx->chain[6] = CPU_BLAKE2S_IVS[6];
    ctx->chain[7] = CPU_BLAKE2S_IVS[7];

    ctx->pos = (keylen > 0) ? BLAKE2S_BLOCK_LENGTH : 0;
  }

  __constant__ uint8_t BLAKE2S_SIGMA[10][16] = {
    {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15}, {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
    {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4}, {7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8},
    {9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13}, {2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9},
    {12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11}, {13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10},
    {6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5}, {10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0}};

  __inline__ __device__ uint32_t cuda_blake2s_ROTR32(uint32_t a, uint8_t b) { return (a >> b) | (a << (32 - b)); }

  __device__ void
  cuda_blake2s_G(cuda_blake2s_ctx_t* ctx, uint32_t m1, uint32_t m2, int32_t a, int32_t b, int32_t c, int32_t d)
  {
    ctx->state[a] = ctx->state[a] + ctx->state[b] + m1;
    ctx->state[d] = cuda_blake2s_ROTR32(ctx->state[d] ^ ctx->state[a], 16);
    ctx->state[c] = ctx->state[c] + ctx->state[d];
    ctx->state[b] = cuda_blake2s_ROTR32(ctx->state[b] ^ ctx->state[c], 12);
    ctx->state[a] = ctx->state[a] + ctx->state[b] + m2;
    ctx->state[d] = cuda_blake2s_ROTR32(ctx->state[d] ^ ctx->state[a], 8);
    ctx->state[c] = ctx->state[c] + ctx->state[d];
    ctx->state[b] = cuda_blake2s_ROTR32(ctx->state[b] ^ ctx->state[c], 7);
  }

  __device__ __forceinline__ void cuda_blake2s_init_state(cuda_blake2s_ctx_t* ctx)
  {
    memcpy(ctx->state, ctx->chain, BLAKE2S_CHAIN_LENGTH);
    ctx->state[8] = BLAKE2S_IVS[0];
    ctx->state[9] = BLAKE2S_IVS[1];
    ctx->state[10] = BLAKE2S_IVS[2];
    ctx->state[11] = BLAKE2S_IVS[3];
    ctx->state[12] = ctx->t0 ^ BLAKE2S_IVS[4];
    ctx->state[13] = ctx->t1 ^ BLAKE2S_IVS[5];
    ctx->state[14] = ctx->f0 ^ BLAKE2S_IVS[6];
    ctx->state[15] = BLAKE2S_IVS[7];
  }

  __device__ __forceinline__ void cuda_blake2s_core(cuda_blake2s_ctx_t* ctx, const uint32_t* m)
  {
    for (int round = 0; round < BLAKE2S_ROUNDS; round++) {
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][0]], m[BLAKE2S_SIGMA[round][1]], 0, 4, 8, 12);
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][2]], m[BLAKE2S_SIGMA[round][3]], 1, 5, 9, 13);
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][4]], m[BLAKE2S_SIGMA[round][5]], 2, 6, 10, 14);
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][6]], m[BLAKE2S_SIGMA[round][7]], 3, 7, 11, 15);
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][8]], m[BLAKE2S_SIGMA[round][9]], 0, 5, 10, 15);
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][10]], m[BLAKE2S_SIGMA[round][11]], 1, 6, 11, 12);
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][12]], m[BLAKE2S_SIGMA[round][13]], 2, 7, 8, 13);
      cuda_blake2s_G(ctx, m[BLAKE2S_SIGMA[round][14]], m[BLAKE2S_SIGMA[round][15]], 3, 4, 9, 14);
    }

    for (int offset = 0; offset < BLAKE2S_CHAIN_SIZE; offset++)
      ctx->chain[offset] = ctx->chain[offset] ^ ctx->state[offset] ^ ctx->state[offset + 8];
  }

  __device__ __forceinline__ void cuda_blake2s_compress(cuda_blake2s_ctx_t* ctx, const BYTE* in, WORD inoffset)
  {
    cuda_blake2s_init_state(ctx);
    uint32_t m[16] = {0};
    memcpy((void*)&m, in + inoffset, BLAKE2S_BLOCK_LENGTH);

    cuda_blake2s_core(ctx, m);
  }

  __device__ void cuda_blake2s_init(cuda_blake2s_ctx_t* ctx, BYTE* key, WORD keylen, WORD digestbitlen)
  {
    memset(ctx, 0, sizeof(cuda_blake2s_ctx_t));
    ctx->keylen = keylen;
    ctx->digestlen = digestbitlen >> 3;
    ctx->pos = 0;
    ctx->t0 = 0;
    ctx->t1 = 0;
    ctx->f0 = 0;
    ctx->chain[0] = BLAKE2S_IVS[0] ^ (ctx->digestlen | (ctx->keylen << 8) | 0x1010000);
    ctx->chain[1] = BLAKE2S_IVS[1];
    ctx->chain[2] = BLAKE2S_IVS[2];
    ctx->chain[3] = BLAKE2S_IVS[3];
    ctx->chain[4] = BLAKE2S_IVS[4];
    ctx->chain[5] = BLAKE2S_IVS[5];
    ctx->chain[6] = BLAKE2S_IVS[6];
    ctx->chain[7] = BLAKE2S_IVS[7];

    if (keylen > 0) {
      memcpy(ctx->buff, key, keylen);
      memcpy(ctx->key, key, keylen);
    }
    ctx->pos = (keylen > 0) ? BLAKE2S_BLOCK_LENGTH : 0;
  }

  __device__ void cuda_blake2s_update(cuda_blake2s_ctx_t* ctx, const BYTE* in, LONG inlen)
  {
    if (inlen == 0) return;

    WORD start = 0;
    int64_t in_index = 0, block_index = 0;

    if (ctx->pos) {
      start = BLAKE2S_BLOCK_LENGTH - ctx->pos;
      if (start < inlen) {
        memcpy(ctx->buff + ctx->pos, in, start);
        ctx->t0 += BLAKE2S_BLOCK_LENGTH;

        if (ctx->t0 == 0) ctx->t1++;

        cuda_blake2s_compress(ctx, ctx->buff, 0);
        ctx->pos = 0;
        memset(ctx->buff, 0, BLAKE2S_BLOCK_LENGTH);
      } else {
        memcpy(ctx->buff + ctx->pos, in, inlen);
        ctx->pos += inlen;
        return;
      }
    }

    block_index = inlen - BLAKE2S_BLOCK_LENGTH;
    for (in_index = start; in_index < block_index; in_index += BLAKE2S_BLOCK_LENGTH) {
      ctx->t0 += BLAKE2S_BLOCK_LENGTH;
      if (ctx->t0 == 0) ctx->t1++;

      cuda_blake2s_compress(ctx, in, in_index);
    }

    memcpy(ctx->buff, in + in_index, inlen - in_index);
    ctx->pos += inlen - in_index;
  }

  __device__ void cuda_blake2s_final(cuda_blake2s_ctx_t* ctx, BYTE* out)
  {
    ctx->f0 = 0xFFFFFFFFUL;
    ctx->t0 += ctx->pos;
    if (ctx->pos > 0 && ctx->t0 == 0) ctx->t1++;

    cuda_blake2s_compress(ctx, ctx->buff, 0);
    memset(ctx->buff, 0, BLAKE2S_BLOCK_LENGTH);
    memset(ctx->state, 0, BLAKE2S_STATE_LENGTH);

    int i4 = 0;
    for (int i = 0; i < BLAKE2S_CHAIN_SIZE && ((i4 = i * 4) < ctx->digestlen); i++) {
      BYTE* BYTEs = (BYTE*)(&ctx->chain[i]);
      if (i4 < ctx->digestlen - 4)
        memcpy(out + i4, BYTEs, 4);
      else
        memcpy(out + i4, BYTEs, ctx->digestlen - i4);
    }
  }

  __device__ void cuda_blake2s_hash_2d(
    CUDA_BLAKE2S_CTX* ctx, const Matrix<BYTE>* inputs, unsigned int number_of_inputs, uint64_t row_idx)
  {
    uint32_t m[BLAKE2S_STATE_SIZE] = {0};
    unsigned int index = 0;
    for (int i = 0; i < number_of_inputs; i++) {
      const Matrix<BYTE>* input = inputs + i;
      for (int j = 0; j < (input->width / 4); j++) {
        m[index] = ((uint32_t*)input->values)[row_idx * (input->width / 4) + j];
        index++;
        if (index == BLAKE2S_STATE_SIZE) {
          cuda_blake2s_init_state(ctx);
          cuda_blake2s_core(ctx, m);
          index = 0;
        }
      }
    }

    if (index) {
      for (int i = index; i < BLAKE2S_STATE_SIZE; i++) {
        m[i] = 0;
      }
      cuda_blake2s_init_state(ctx);
      cuda_blake2s_core(ctx, m);
    }
  }

  __global__ void
  kernel_blake2s_hash(const BYTE* indata, WORD inlen, BYTE* outdata, WORD n_batch, WORD BLAKE2S_BLOCK_SIZE, bool mmcs)
  {
    WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread >= n_batch) { return; }
    BYTE key[32] = ""; // Null key
    WORD keylen = 0;
    CUDA_BLAKE2S_CTX blake_ctx;
    const BYTE* in = indata + thread * inlen;
    BYTE* out = outdata + thread * BLAKE2S_BLOCK_SIZE;

    cuda_blake2s_init(&blake_ctx, key, keylen, (BLAKE2S_BLOCK_SIZE << 3));
    if (mmcs) {
      memset(blake_ctx.chain, 0, BLAKE2S_CHAIN_LENGTH);
      cuda_blake2s_compress(&blake_ctx, in, 0);
      memcpy(out, blake_ctx.chain, BLAKE2S_CHAIN_LENGTH);
    } else {
      cuda_blake2s_update(&blake_ctx, in, inlen);
      cuda_blake2s_final(&blake_ctx, out);
    }
  }

  __global__ void
  hash_2d_kernel(const Matrix<BYTE>* inputs, BYTE* output, unsigned int number_of_inputs, unsigned int output_len)
  {
    uint64_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= inputs[0].height) { return; }

    CUDA_BLAKE2S_CTX ctx;
    cuda_blake2s_init(&ctx, nullptr, 0, (32 << 3));
    memset(ctx.chain, 0, BLAKE2S_CHAIN_LENGTH);
    cuda_blake2s_hash_2d(&ctx, inputs, number_of_inputs, idx);
    memcpy(output + idx * output_len, ctx.chain, BLAKE2S_CHAIN_LENGTH);
  }

  __global__ void compress_and_inject_kernel(
    const Matrix<BYTE>* matrices_to_inject, unsigned int number_of_inputs, const BYTE* prev_layer, BYTE* next_layer)
  {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint64_t number_of_rows = d_next_pow_of_two(matrices_to_inject[0].height);
    if (idx >= number_of_rows) { return; }

    CUDA_BLAKE2S_CTX ctx;
    cuda_blake2s_init(&ctx, nullptr, 0, (32 << 3));
    memset(ctx.chain, 0, BLAKE2S_CHAIN_LENGTH);

    size_t next_layer_len = matrices_to_inject[0].height;
    cuda_blake2s_compress(&ctx, &prev_layer[idx * BLAKE2S_BLOCK_LENGTH], 0);

    if (idx < next_layer_len) { cuda_blake2s_hash_2d(&ctx, matrices_to_inject, number_of_inputs, idx); }

    memcpy(next_layer + idx * BLAKE2S_CHAIN_LENGTH, ctx.chain, BLAKE2S_CHAIN_LENGTH);
  }

  extern "C" {
  hipError_t
  cuda_blake2s_hash_batch(BYTE* key, WORD keylen, BYTE* in, WORD inlen, BYTE* out, WORD output_len, WORD n_batch)
  {
    BYTE* cuda_indata;
    BYTE* cuda_outdata;
    const WORD BLAKE2S_BLOCK_SIZE = output_len;
    hipMalloc(&cuda_indata, inlen * n_batch);
    hipMalloc(&cuda_outdata, BLAKE2S_BLOCK_SIZE * n_batch);
    assert(keylen <= 32);

    hipMemcpy(cuda_indata, in, inlen * n_batch, hipMemcpyHostToDevice);

    WORD thread = 256;
    WORD block = (n_batch + thread - 1) / thread;
    kernel_blake2s_hash<<<block, thread>>>(cuda_indata, inlen, cuda_outdata, n_batch, BLAKE2S_BLOCK_SIZE, false);
    hipMemcpy(out, cuda_outdata, BLAKE2S_BLOCK_SIZE * n_batch, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // hipError_t error = hipGetLastError();
    // if (error != hipSuccess) { printf("Error cuda blake2s hash: %s \n", hipGetErrorString(error)); }
    hipFree(cuda_indata);
    hipFree(cuda_outdata);
    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }
  }

  hipError_t Blake2s::run_hash_many_kernel(
    const BYTE* input,
    BYTE* output,
    WORD number_of_states,
    WORD input_len,
    WORD output_len,
    const device_context::DeviceContext& ctx) const
  {
    const WORD BLAKE2S_BLOCK_SIZE = output_len;
    WORD thread = 256;
    WORD block = (number_of_states + thread - 1) / thread;

    kernel_blake2s_hash<<<block, thread, 0, ctx.stream>>>(
      input, input_len, output, number_of_states, BLAKE2S_BLOCK_SIZE, !use_iv);

    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }

  hipError_t Blake2s::hash_2d(
    const Matrix<BYTE>* inputs,
    BYTE* states,
    unsigned int number_of_inputs,
    unsigned int output_len,
    uint64_t number_of_rows,
    const device_context::DeviceContext& ctx) const
  {
    const WORD BLAKE2S_BLOCK_SIZE = output_len;
    WORD thread = 256;
    WORD block = (number_of_rows + thread - 1) / thread;

    hash_2d_kernel<<<block, thread, 0, ctx.stream>>>(inputs, states, number_of_inputs, output_len);
    return CHK_LAST();
  }

  hipError_t Blake2s::compress_and_inject(
    const Matrix<BYTE>* matrices_to_inject,
    unsigned int number_of_inputs,
    uint64_t number_of_rows,
    const BYTE* prev_layer,
    BYTE* next_layer,
    unsigned int digest_elements,
    const device_context::DeviceContext& ctx) const
  {
    const WORD BLAKE2S_BLOCK_SIZE = digest_elements;
    WORD thread = 256;
    WORD block = (number_of_rows + thread - 1) / thread;

    compress_and_inject_kernel<<<block, thread, 0, ctx.stream>>>(
      matrices_to_inject, number_of_inputs, prev_layer, next_layer);
    return CHK_LAST();
  }

} // namespace blake2s