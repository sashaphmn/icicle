#include <chrono>
#include "gpu-utils/device_context.cuh"

#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <iomanip>
#include "extern.cu"

using namespace blake2s;

#define START_TIMER(timer) auto timer##_start = std::chrono::high_resolution_clock::now();
#define END_TIMER(timer, msg)                                                                                          \
  printf("%s: %.0f us\n", msg, FpMicroseconds(std::chrono::high_resolution_clock::now() - timer##_start).count());

// extern "C" {
// void cuda_blake2s_hash_batch(BYTE* key, WORD keylen, BYTE* in, WORD inlen, BYTE* out, WORD n_outbit, WORD n_batch);
// }

void print_hash(BYTE* hash, WORD len)
{
  printf("Hash Len: %d \n", len);
  printf("BLAKE2S hash:\n");
  for (WORD i = 0; i < len; i++) {
    printf("%02x", hash[i]);
  }
  printf("\n");
}

BYTE* read_file(const char* filename, size_t* filesize)
{
  FILE* file = fopen(filename, "rb");
  if (!file) {
    perror("Failed to open file");
    exit(EXIT_FAILURE);
  }

  fseek(file, 0, SEEK_END);
  *filesize = ftell(file);
  fseek(file, 0, SEEK_SET);

  BYTE* buffer = (BYTE*)malloc(*filesize);
  if (!buffer) {
    perror("Failed to allocate memory");
    fclose(file);
    exit(EXIT_FAILURE);
  }

  size_t bytesRead = fread(buffer, 1, *filesize, file);
  if (bytesRead != *filesize) {
    perror("Failed to read file");
    free(buffer);
    fclose(file);
    exit(EXIT_FAILURE);
  }

  fclose(file);
  return buffer;
}

int main(int argc, char** argv)
{
  using FpMilliseconds = std::chrono::duration<float, std::chrono::milliseconds::period>;
  using FpMicroseconds = std::chrono::duration<float, std::chrono::microseconds::period>;

  BYTE* input;
  size_t inlen;
  const char* input_filename;
  const char* default_input = "aaaaaaaaaaa";

  if (argc < 2) {
    // Use default input if no file is provided
    input = (BYTE*)default_input;
    inlen = strlen(default_input);
  } else {
    input_filename = argv[1];
    input = read_file(input_filename, &inlen);
  }

  // Test parameters
  BYTE key[32] = ""; // Example key
  WORD keylen = strlen((char*)key);
  WORD n_outbit = 256; // Output length in bits
  WORD n_batch = 1;    // Number of hashes to compute in parallel
  // Allocate memory for the output
  WORD outlen = n_outbit / 8;
  BYTE* output = (BYTE*)malloc(outlen * n_batch);
  if (!output) {
    perror("Failed to allocate memory for output");
    if (argc >= 2) free(input); // Free file buffer if it was allocated
    return EXIT_FAILURE;
  }

  printf("Key len: %d \n", keylen);

  // Perform the hashing
  START_TIMER(blake_timer)
  HashConfig config = default_hash_config();

  blake2s_cuda(input, output, n_batch, inlen, outlen, config);
  END_TIMER(blake_timer, "Blake Timer")

  // Print the result
  print_hash(output, outlen);

  // Clean up
  free(output);
  if (argc >= 2) free(input); // Free file buffer if it was allocated
  return 0;
}
