#include "hip/hip_runtime.h"
#include "fields/field_config.cuh"
using namespace field_config;

#include "fri.cu"
#include "utils/utils.h"
#include "fields/point.cuh"

namespace fri {
  /**
   * Extern "C" version of [fold_line](@ref fold_line) function with the following values of
   * template parameters (where the field is given by `-DFIELD` env variable during build):
   *  - `E` is the extension field type used for evaluations and alpha
   *  - `S` is the scalar field type used for domain elements
   * @param line_eval Pointer to the array of evaluations on the line
   * @param domain_elements Pointer to the array of domain elements
   * @param alpha The folding factor
   * @param folded_evals Pointer to the array where folded evaluations will be stored
   * @param n The number of evaluations
   * @param ctx The device context; if the stream is not 0, then everything is run async
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_line)(
    q_extension_t* line_eval,
    scalar_t* domain_elements,
    q_extension_t alpha,
    q_extension_t* folded_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    return fri::fold_line(line_eval, domain_elements, alpha, folded_evals, n, cfg);
  };

  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_line_new)(
    q_extension_t* line_eval,
    uint64_t line_domain_initial_index,
    uint32_t line_domain_log_size,
    q_extension_t alpha,
    q_extension_t* folded_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    circle_math::LineDomain<fp_config, scalar_t> line_domain = circle_math::LineDomain<fp_config, scalar_t>(line_domain_initial_index, line_domain_log_size);
    scalar_t* domain_elements;
    auto size = line_domain.size();

    CHK_IF_RETURN(hipMalloc(&domain_elements, size));

    uint64_t num_threads = max(1, min(unsigned(size), 256));
    uint64_t num_blocks = (size + num_threads - 1) / num_threads;
    fri::get_line_domain_values<<<num_blocks, num_threads>>>(line_domain, domain_elements, size);

    cfg.are_domain_elements_on_device = true;
    return fri::fold_line(line_eval, domain_elements, alpha, folded_evals, n, cfg);
  };

  /**
   * Extern "C" version of [fold_circle_into_line](@ref fold_circle_into_line) function with the following values of
   * template parameters (where the field is given by `-DFIELD` env variable during build):
   *  - `E` is the extension field type used for evaluations and alpha
   *  - `S` is the scalar field type used for domain elements
   * @param circle_evals Pointer to the array of evaluations on the circle
   * @param domain_elements Pointer to the array of domain elements
   * @param alpha The folding factor
   * @param folded_line_evals Pointer to the array where folded evaluations will be stored
   * @param n The number of evaluations
   * @param ctx The device context; if the stream is not 0, then everything is run async
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_circle_into_line)(
    q_extension_t* circle_evals,
    scalar_t* domain_elements,
    q_extension_t alpha,
    q_extension_t* folded_line_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    return fri::fold_circle_into_line(circle_evals, domain_elements, alpha, folded_line_evals, n, cfg);
  };

  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_circle_into_line_new)(
    q_extension_t* circle_evals,
    uint64_t domain_initial_index,
    uint32_t domain_log_size,
    q_extension_t alpha,
    q_extension_t* folded_line_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    domain_t domain(coset_t(domain_initial_index, domain_log_size));
    scalar_t* domain_elements;
    auto size = domain.size();

    CHK_IF_RETURN(hipMalloc(&domain_elements, size));

    uint64_t num_threads = max(1, min(unsigned(size), 256));
    uint64_t num_blocks = (size + num_threads - 1) / num_threads;
    fri::get_circle_domain_ys<<<num_blocks, num_threads>>>(domain, domain_elements, size);

    cfg.are_domain_elements_on_device = true;

    return fri::fold_circle_into_line(circle_evals, domain_elements, alpha, folded_line_evals, n, cfg);
  };
} // namespace fri
